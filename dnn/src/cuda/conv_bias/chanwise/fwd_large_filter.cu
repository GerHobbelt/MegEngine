/**
 * \file dnn/src/cuda/conv_bias/chanwise/fwd_depthwise_large_filter.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */

#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"
// #include "src/cuda/conv_bias/chanwise/fwd_depthwise_large_filter.cuh"
#include "src/cuda/conv_bias/chanwise/kern.cuh"
#include "src/cuda/conv_bias/chanwise/kern_helper.cuh"
#include "src/cuda/conv_bias/chanwise/launch_config.cuh"
#include "src/cuda/fp16_help.cuh"

using namespace megdnn;
using namespace cuda;
using namespace conv_bias;
using namespace chanwise;

#include "src/cuda/conv_bias/chanwise/depthwise_large_filter_algo.inl"

namespace megdnn {
namespace cuda {
namespace conv_bias {
namespace chanwise {

// =====================================fwd=====================================

#define check

template <>
void run_fwd_depthwise_large_filter(
        float* dst, const float* src, const float* flt, const Param& param,
        hipStream_t stream) {
    INSTANCE(DepthwiseConv2dDirection::DIRECTION_FORWARD)
}

}  // namespace chanwise
}  // namespace conv_bias
}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cuda.doxygen
